#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

extern "C" {
    #include "common.h"
}

/*
static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}
*/

extern "C"
int GuassianEliminationV1 (float** inputMatrix, int rows, int cols) {
    float *hostMatrix = 0;
	float *deviceMatrix = 0;
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

    int i, j;

    hostMatrix = (float *)malloc(rows * cols * sizeof(float));
    if (!hostMatrix) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            hostMatrix[IDX2C(i,j,rows)] = inputMatrix[i][j];
        }
    }
    
    cudaStat = hipMalloc ((void**) &deviceMatrix, rows * cols * sizeof(*hostMatrix));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix (rows, cols, sizeof(*hostMatrix), hostMatrix, rows, deviceMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed\n");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }






    //If you are gonna use this command, uncomment this function "modify" at the top of this file, else it won't work
    //modify (handle, deviceMatrix, rows, cols, 1, 2, 16.0f, 12.0f);    

    float scalar = 10.0f;
    hipblasSscal (handle, rows, &scalar, &deviceMatrix[IDX2C(0,2,rows)], 1);




    stat = hipblasGetMatrix (rows, cols, sizeof(*hostMatrix), deviceMatrix, rows, hostMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data upload failed");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipDeviceSynchronize();

    hipFree (deviceMatrix);
    hipblasDestroy(handle);

    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            inputMatrix[i][j] = hostMatrix[IDX2C(i,j,rows)];
        }
    }

    free(hostMatrix);

    return 0;
}