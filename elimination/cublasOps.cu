#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "common.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int GuassianEliminationV1 (float** inputMatrix, int rows, int cols) {
    float *hostMatrix = 0;
	float *deviceMatrix = 0;
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

    int i, j;

    hostMatrix = (float *)malloc(rows * cols * sizeof(float));
    if (!hostMatrix) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            hostMatrix[IDX2C(i,j,rows)] = inputMatrix[i][j];
        }
    }
    
    printCublasMatrixArray(hostMatrix, (rows * cols));
    
    cudaStat = hipMalloc ((void**) &deviceMatrix, rows * cols * sizeof(*hostMatrix));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix (rows, cols, sizeof(*hostMatrix), hostMatrix, rows, deviceMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed\n");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }





    //Code Here





    stat = hipblasGetMatrix (rows, cols, sizeof(*hostMatrix), deviceMatrix, rows, hostMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data upload failed");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipFree (deviceMatrix);
    hipblasDestroy(handle);

    //Do something with the host matrix, print it or whatever
    printCublasMatrixArray(hostMatrix, (rows * cols));

    free(hostMatrix);

    return 0;
}