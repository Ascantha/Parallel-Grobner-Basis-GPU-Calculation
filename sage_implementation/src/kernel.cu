#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void __global__ kernel_add_one(int* a, int length) {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    while(gid < length) {
    	a[gid] += 1;
        gid += blockDim.x*gridDim.x;
    }
}

int Inverse_Integer_Mod (int x, int n) {
    int tmp = 0;
    int a = 0;
    int b = 0;
    int last_t = 0;
    int t = 0;
    int next_t = 0;
    int q = 0;

    if (n == 1) {
        return 0;
    }

    a = n;
    b = x;
    t = 0;
    next_t = 1;

    while (b != 0) {
        if (b == 1) {
            next_t = next_t % n;
            if (next_t < 0) {
                next_t = next_t + n;
            }
            return next_t;
        }

        q = a / b;
        tmp = b;
        b = a % b;
        a = tmp;
        last_t = t;
        t = next_t;
        next_t = last_t - q * t;
    }

    return 0;
}

int __device__ Add_Integer_Mod (int n, int m, int p) {
    return (n + m) % p;
}

int __device__ Mul_Integer_Mod (int n, int m, int p) {
    return (n * m) % p;
}

//Call: intModDAXPY <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, devicematrix*, rows, field_size);
void __global__ intModDAXPY (int n, int scalar, double* x, double* y, int inc, int p) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < n) {
        int j = i * inc;

        int aX = Mul_Integer_Mod(scalar, (int)x[j], p);
        y[j] = (double)Add_Integer_Mod(aX, (int)y[j], p);
    }
}

//Call: intModDScale <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, rows, field_size)
void __global__ intModDScale (int n, int scalar, double* x, int inc, int p) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < n) {
        int j = i * inc;

        x[j] = (double)Mul_Integer_Mod(scalar, x[j], p);
    }
}

int F4_5_GuassianElimination_Finite (double * inputMatrix, int rows, int cols, int dontPrint, int checkRef, int field_size) {
    double *hostMatrix = 0;
    double *deviceMatrix = 0;
    hipError_t cudaStat;
	hipblasStatus_t stat;
    hipblasHandle_t handle;

    int* rPiv;
    int i, j, c, r;

    rPiv = (int *)malloc(cols * sizeof(int));

    if(dontPrint == 0) {
        printf("F4-5 Guassian Elimination\n");
        printf("====================================================================================================================================\n");
        printf("                                                        Start Algorithm\n");
        printf("====================================================================================================================================\n");
    }
    
    for (i = 0; i < cols; i++) {
        rPiv[i] = -1;
    }

    //Initialize Array for the Host Matrix
    hostMatrix = (double *)malloc(rows * cols * sizeof(double));
    if (!hostMatrix) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            hostMatrix[IDX2C(i,j,rows)] = inputMatrix[IDX2C(i,j,rows)];
        }
    }

    //Allocate memory for Device Matrix Array
    cudaStat = hipMalloc ((void**) &deviceMatrix, rows * cols * sizeof(*hostMatrix));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    //Initialize CuBLAS object
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    //Push data to the Device
    stat = hipblasSetMatrix (rows, cols, sizeof(*hostMatrix), hostMatrix, rows, deviceMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed\n");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    double *tempVector = (double *)malloc(rows* sizeof(double));
    double *tempAxpyScal = (double *) malloc (sizeof(double));
    double *inverseRounder = (double *)malloc(sizeof(double));
    int int_scalar = 0;
    *inverseRounder = 1;

    for (c = 0; c < cols; c++) {
        //Download the Vector
        stat = hipblasGetVector(rows, sizeof(double), &deviceMatrix[IDX2C(0,c,rows)], 1, tempVector, 1);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("Data Vector download failed");
            hipFree (deviceMatrix);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
        
        for (r = 0; r < rows; r++) {
            if(tempVector[r] != 0.0f && rPiv[r] == -1) {
                rPiv[r] = r;

                int_scalar = Inverse_Integer_Mod((int)tempVector[r], field_size);
                //intModDScale <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, rows, field_size)
                intModDScale<<<(cols+255)/256, 256>>>(cols, int_scalar, &deviceMatrix[IDX2C(r,0,rows)], rows, field_size);
                hipDeviceSynchronize();

                for (i = r + 1; i < rows; i++) {
                    if (tempVector[i] != 0.0f) {
                        int_scalar = field_size - (int)tempVector[i];

                        //intModDAXPY <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, devicematrix*, rows, field_size);
                        intModDAXPY<<<(cols+255)/256, 256>>>(cols, int_scalar, &deviceMatrix[IDX2C(r,0,rows)], &deviceMatrix[IDX2C(i,0,rows)], rows, field_size);
                        hipDeviceSynchronize();
                    }
                }

                break;
            }
        }
    }

    //Download Matrix from the Device -> Host
    stat = hipblasGetMatrix (rows, cols, sizeof(*hostMatrix), deviceMatrix, rows, hostMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data upload failed");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    //Sync up the device
    hipDeviceSynchronize();

    if (checkRef == 1) {
        printf("Checking if NaN/Infinite rows are present...\n");
    }
    
    //Bring data back to input matrix to be passed back to f4/5 algorithm
    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            inputMatrix[IDX2C(i,j,rows)] = hostMatrix[IDX2C(i,j,rows)];
        }
    }

    //Free all the memory used
    hipFree (deviceMatrix);
    hipblasDestroy(handle);

    free(hostMatrix);
    free(rPiv);
    free(inverseRounder);
    free(tempVector);

    return 0;
}

double __device__ Add_Integer_Mod_Double (double n, double m, int p) {
    return fmod((n + m), (double)p);
}

double __device__ Mul_Integer_Mod_Double (double n, double m, int p) {
    return fmod((n * m), (double)p);
}

//Call: intModDAXPY <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, devicematrix*, rows, field_size);
void __global__ intModDAXPY_Double (int n, double scalar, double* x, double* y, int inc, int p) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < n) {
        int j = i * inc;

        double aX = Mul_Integer_Mod_Double(scalar, x[j], p);
        y[j] = Add_Integer_Mod_Double(aX, y[j], p);
    }
}

//Call: intModDScale <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, rows, field_size)
void __global__ intModDScale_Double (int n, double scalar, double* x, int inc, int p) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < n) {
        int j = i * inc;

        x[j] = Mul_Integer_Mod_Double(scalar, x[j], p);
    }
}

int F4_5_GuassianElimination_Finite_Double (double * inputMatrix, int rows, int cols, int dontPrint, int checkRef, int field_size) {
    double *hostMatrix = 0;
    double *deviceMatrix = 0;
    hipError_t cudaStat;
	hipblasStatus_t stat;
    hipblasHandle_t handle;

    int* rPiv;
    int i, j, c, r;

    rPiv = (int *)malloc(cols * sizeof(int));

    if(dontPrint == 0) {
        printf("F4-5 Guassian Elimination\n");
        printf("====================================================================================================================================\n");
        printf("                                                        Start Algorithm\n");
        printf("====================================================================================================================================\n");
    }
    
    for (i = 0; i < cols; i++) {
        rPiv[i] = -1;
    }

    //Initialize Array for the Host Matrix
    hostMatrix = (double *)malloc(rows * cols * sizeof(double));
    if (!hostMatrix) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    //Copy data to Host Matrix
    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            hostMatrix[IDX2C(i,j,rows)] = inputMatrix[IDX2C(i,j,rows)];
        }
    }

    //Allocate memory for Device Matrix Array
    cudaStat = hipMalloc ((void**) &deviceMatrix, rows * cols * sizeof(*hostMatrix));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    //Initialize CuBLAS object
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    //Push data to the Device
    stat = hipblasSetMatrix (rows, cols, sizeof(*hostMatrix), hostMatrix, rows, deviceMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed\n");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    double *tempVector = (double *)malloc(rows* sizeof(double));
    double *tempAxpyScal = (double *) malloc (sizeof(double));
    double double_scalar = 0.0f;

    for (c = 0; c < cols; c++) {
        //Download the Vector
        stat = hipblasGetVector(rows, sizeof(double), &deviceMatrix[IDX2C(0,c,rows)], 1, tempVector, 1);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("Data Vector download failed");
            hipFree (deviceMatrix);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
        
        for (r = 0; r < rows; r++) {
            if(tempVector[r] != 0.0f && rPiv[r] == -1) {
                rPiv[r] = r;

                double_scalar = (double)Inverse_Integer_Mod(tempVector[r], field_size);
                //intModDScale <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, rows, field_size)
                intModDScale_Double<<<(cols+255)/256, 256>>>(cols, double_scalar, &deviceMatrix[IDX2C(r,0,rows)], rows, field_size);
                hipDeviceSynchronize();

                for (i = r + 1; i < rows; i++) {
                    if (tempVector[i] != 0.0f) {
                        double_scalar = field_size - tempVector[i];

                        //intModDAXPY <<<(N+255)/256, 256>>> (cols, scalar, devicematrix*, devicematrix*, rows, field_size);
                        intModDAXPY_Double<<<(cols+255)/256, 256>>>(cols, double_scalar, &deviceMatrix[IDX2C(r,0,rows)], &deviceMatrix[IDX2C(i,0,rows)], rows, field_size);
                        hipDeviceSynchronize();
                    }
                }

                break;
            }
        }
    }

    //Download Matrix from the Device -> Host
    stat = hipblasGetMatrix (rows, cols, sizeof(*hostMatrix), deviceMatrix, rows, hostMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data upload failed");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    //Sync up the device
    hipDeviceSynchronize();

    if (checkRef == 1) {
        printf("Checking if NaN/Infinite rows are present...\n");
    }
    
    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            inputMatrix[IDX2C(i,j,rows)] = hostMatrix[IDX2C(i,j,rows)];
        }
    }

    //Free all the memory used
    hipFree (deviceMatrix);
    hipblasDestroy(handle);

    free(hostMatrix);
    free(rPiv);
    free(tempVector);

    return 0;
}

int F4_5_GuassianElimination (double * inputMatrix, int rows, int cols, int dontPrint, int checkRef) {
    printf("TEST\n");
    double *hostMatrix = 0;
    double *deviceMatrix = 0;
    hipError_t cudaStat;
	hipblasStatus_t stat;
    hipblasHandle_t handle;

    int* rPiv;
    int i, j, c, r;

    rPiv = (int *)malloc(cols * sizeof(int));

    if(dontPrint == 0) {
        printf("F4-5 Guassian Elimination\n");
        printf("====================================================================================================================================\n");
        printf("                                                        Start Algorithm\n");
        printf("====================================================================================================================================\n");
    }
    
    for (i = 0; i < cols; i++) {
        rPiv[i] = -1;
    }

    //Initialize Array for the Host Matrix
    hostMatrix = (double *)malloc(rows * cols * sizeof(double));
    if (!hostMatrix) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            hostMatrix[IDX2C(i,j,rows)] = inputMatrix[IDX2C(i,j,rows)];
        }
    }

    //Allocate memory for Device Matrix Array
    cudaStat = hipMalloc ((void**) &deviceMatrix, rows * cols * sizeof(*hostMatrix));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    //Initialize CuBLAS object
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    //Push data to the Device
    stat = hipblasSetMatrix (rows, cols, sizeof(*hostMatrix), hostMatrix, rows, deviceMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed\n");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    double *tempVector = (double *)malloc(rows* sizeof(double));
    double *tempAxpyScal = (double *) malloc (sizeof(double));
    double *inverseRounder = (double *)malloc(sizeof(double));
    double scalar = 0.0f;

    *inverseRounder = 1;

    for (c = 0; c < cols; c++) {
        //Download the Vector
        stat = hipblasGetVector(rows, sizeof(double), &deviceMatrix[IDX2C(0,c,rows)], 1, tempVector, 1);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("Data Vector download failed");
            hipFree (deviceMatrix);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
        
        for (r = 0; r < rows; r++) {
            if(tempVector[r] != 0.0f && rPiv[r] == -1) {
                rPiv[r] = r;

                scalar = tempVector[r];    
                scalar = powf(scalar, -1); 
                
                stat = hipblasDscal (handle, cols, &scalar, &deviceMatrix[IDX2C(r,0,rows)], rows);
                if (stat != HIPBLAS_STATUS_SUCCESS) {
                    printf ("Device operation failed (row scalar * inverse of leading term)\n");
                    return EXIT_FAILURE;
                }  
                             
                
                //Copy 1 to location where the LT should be 1 because floats/doubles are not accurate enough
                hipMemcpy(&deviceMatrix[IDX2C(r,c,rows)], inverseRounder, sizeof(double), hipMemcpyHostToDevice);

                for (i = r + 1; i < rows; i++) {
                    if (tempVector[i] != 0.0f) {
                        *tempAxpyScal = -(tempVector[i]);
                        stat = hipblasDaxpy(handle, cols, tempAxpyScal, &deviceMatrix[IDX2C(r,0,rows)], rows, &deviceMatrix[IDX2C(i,0,rows)], rows);
                        if (stat != HIPBLAS_STATUS_SUCCESS) {
                            printf ("Device operation failed (dAxpy)\n");
                            return EXIT_FAILURE;
                        }
                    }
                }

                break;
            }
        }
    }

    //Download Matrix from the Device -> Host
    stat = hipblasGetMatrix (rows, cols, sizeof(*hostMatrix), deviceMatrix, rows, hostMatrix, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data upload failed");
        hipFree (deviceMatrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    //Sync up the device
    hipDeviceSynchronize();

    if (checkRef == 1) {
        printf("Checking if NaN/Infinite rows are present...\n");
    }
    
    for(j = 0; j < cols; j++) {
        for(i = 0; i < rows; i++) {
            inputMatrix[IDX2C(i,j,rows)] = hostMatrix[IDX2C(i,j,rows)];
        }
    }

    //Free all the memory used
    hipFree (deviceMatrix);
    hipblasDestroy(handle);

    free(hostMatrix);
    free(rPiv);
    free(inverseRounder);
    free(tempVector);

    return 0;
}